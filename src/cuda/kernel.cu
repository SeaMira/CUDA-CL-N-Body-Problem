#include "hip/hip_runtime.h"
#include "kernel.cuh"

__device__ void bodyInteraction(float *pos, float *vel, int bodies, float step,int gindex){
	float k = 0;
  // getting this thread's position
  float x = pos[gindex*3];
  float y = pos[gindex*3+1];
  float z = pos[gindex*3+2];

  // getting this thread's velocity
  float vx = vel[gindex*3];
  float vy = vel[gindex*3+1];
  float vz = vel[gindex*3+2];
  while (k < 100 ) {

    // getting new acceleration
    float acc[3] = {0};
    float dx, dy, dz, dist, cubedDist, vec;
    for (int i = 0; i < bodies*3; i+=3) {
      dx = pos[i] - x;
      dy = pos[i+1] - y;
      dz = pos[i+2] - z;

      dist = sqrt(dx*dx + dy*dy + dz*dz);
      if (dist != 0) {
        cubedDist = dist*dist*dist;
        vec = 1/cubedDist;

        acc[0] += vec*dx;
        acc[1] += vec*dy;
        acc[2] += vec*dz;
      }
    }

    vx += acc[0]*step;
    vy += acc[1]*step;
    vz += acc[2]*step;

    x += vx*step;
    y += vy*step;
    z += vz*step;

    __syncthreads();

    pos[gindex*3] = x;
    pos[gindex*3+1] = y;
    pos[gindex*3+2] = z;

    vel[gindex*3] = vx;
    vel[gindex*3+1] = vy;
    vel[gindex*3+2] = vz;

    __syncthreads();
    k += step;
  }
}

__device__ void bodyInteractionLocal(float *pos, float *vel, int bodies, float step,int gindex){
	float k = 0;
  extern __shared__ float sharedmem[];
  int numItems = blockDim.x;
  int tnum = blockIdx.x;
  unsigned int loop = bodies/numItems;
  // getting this thread's position
  float x = pos[gindex*3];
  float y = pos[gindex*3+1];
  float z = pos[gindex*3+2];

  // getting this thread's velocity
  float vx = vel[gindex*3];
  float vy = vel[gindex*3+1];
  float vz = vel[gindex*3+2];
  while (k < 100 ) {
    float acc[3] = {0};
    float dx, dy, dz, dist, cubedDist, vec;
    for(int l=0;l<loop;l++){
      int b_id=numItems*l+tnum;
      if(b_id<bodies){
        sharedmem[tnum*3] = pos[tnum*3 + numItems*l];
        sharedmem[tnum*3+1] = pos[tnum*3 + numItems*l+1];
        sharedmem[tnum*3+2] = pos[tnum*3 + numItems*l+2];
      }
      __syncthreads();
      int range=min(numItems,bodies-numItems*l)*3;
      for(int i=0;i<range;i+=3){
        dx = sharedmem[i] - x;
        dy = sharedmem[i+1] - y;
        dz = sharedmem[i+2] - z;

        dist = sqrt(dx*dx + dy*dy + dz*dz);
        if (dist != 0) {
          cubedDist = dist*dist*dist;
          vec = 1/cubedDist;

          acc[0] += vec*dx;
          acc[1] += vec*dy;
          acc[2] += vec*dz;
        }
      }
    }
    vx += acc[0]*step;
    vy += acc[1]*step;
    vz += acc[2]*step;

    x += vx*step;
    y += vy*step;
    z += vz*step;

    __syncthreads();

    pos[gindex*3] = x;
    pos[gindex*3+1] = y;
    pos[gindex*3+2] = z;

    vel[gindex*3] = vx;
    vel[gindex*3+1] = vy;
    vel[gindex*3+2] = vz;

    __syncthreads();
    k += step;
  }
}

__global__ void bodyInteraction1D(float *pos,float *vel, int bodies,float step){
  int gindex = blockDim.x * blockIdx.x + threadIdx.x;
  if (gindex < bodies) {
    bodyInteraction(pos,vel,bodies,step,gindex);
  }
}
__global__ void bodyInteraction2D(float *pos,float *vel, int bodies,float step){
  int globalIdx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int globalIdx_y = blockIdx.y * blockDim.y + threadIdx.y;
  int gindex=globalIdx_y*blockDim.x*blockIdx.x+globalIdx_x;
  if (gindex<bodies){
    bodyInteraction(pos,vel,bodies,step,gindex);
  }
}

__global__ void bodyInteractionLocal1D(float *pos,float *vel, int bodies,float step){
  int gindex = blockDim.x * blockIdx.x + threadIdx.x;
  if (gindex < bodies) {
    bodyInteractionLocal(pos,vel,bodies,step,gindex);
  }
}